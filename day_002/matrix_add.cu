#include <stdio.h>
#include <random>
#include <hip/hip_runtime.h>


#define CUDA_CHECK(call) \
do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
               hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

__global__ void matrixAdd(float* A, float* B, float* C, int width, int height){
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width){
        C[row * width + col] = A[row * width + col] + B[row * width + col];
    }
}

void print_matrix(float* A, int width, int height){
    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            printf("%.2f", A[i*width+j]);
        }
        printf("\n");
    }
}

int main(){
    int height = 5;
    int width = 6;

    float* h_A = new float[height*width];
    float* h_B = new float[height*width];
    float* h_C = new float[height*width];

    //  initialization

    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            int idx = i * width + j;
            h_A[idx] = 1.0f;
            h_B[idx] = 2.0f;
        }
    }
    printf("Matrix A\n");
    print_matrix(h_A, width, height);
    printf("Matrix B\n");
    print_matrix(h_B, width, height);


    float *d_A, *d_B, *d_C; //device
    size_t size = width * height * sizeof(float);

    // should define error checking here in case OOM
    CUDA_CHECK(hipMalloc(&d_A, size));CUDA_CHECK(hipMalloc(&d_B, size));CUDA_CHECK(hipMalloc(&d_C, size));

    // copy to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    //lauch kernel
    // int threadsPerBlock = 256;
    // int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    dim3 numThreadsPerBlock(16, 16);
    dim3 numBlocks(
        (width + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
        (height + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y
    );
    matrixAdd<<<numBlocks, numThreadsPerBlock >>>(d_A, d_B, d_C, width, height);

    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printf("Matrix C\n");
    print_matrix(h_C, width, height);

    delete[] h_A; delete[] h_B; delete[] h_C;
    hipFree(d_A); hipFree(d_B);hipFree(d_C);

    return 0;
}